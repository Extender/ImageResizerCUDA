#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include "decimal.h"
#include "colordefs.h"
#include "text.h"

#include <iostream>

#define bilinearInterpolate(c00,c01,c10,c11,w1,w2,w3,w4) (w1*c00+w2*c01+w3*c10+w4*c11)

#define CUDA_THREADS_PER_BLOCK 1024 // Optimal value (calculated)

#if 0==1
#define __shared__
#define __global__
#define __device__
#define __host__
#endif

void cudaLog(const char *str)
{
    std::cout<<str<<std::endl;
}

__global__ void device_bilinearResize(int origImgWidth,int origImgHeight,int newImgWidth,decimal_t origRatio,int blocksPerLine,uint32_t *origImgData,uint32_t *newImgData)
{
    // Each block is responsible for 1024 px of a single row. Do not use shared memory, as we cannot predict the distances between two lines we need in the source image.

    int threadId=threadIdx.x;
    int blockId=blockIdx.x;
    int newY=(blockId-(blockId%blocksPerLine))/blocksPerLine;
    int blockIdInLine=blockId-newY*blocksPerLine;
    int newX=blockIdInLine*CUDA_THREADS_PER_BLOCK+threadId;

    if(newX>=newImgWidth)
        return;

    // y-specific:

    decimal_t oldYRF;
    int oldY;
    decimal_t oldYF;
    decimal_t yDiff;
    decimal_t yDiffR;

    oldYRF=origRatio*(decimal_t)newY;
    oldY=(int)floor(oldYRF);
    oldYF=(decimal_t)oldY;
    yDiff=oldYRF-oldYF;
    yDiffR=1.0-yDiff;

    // x-specific:

    decimal_t oldXRF=origRatio*(decimal_t)newX;
    int oldX=(int)floor(oldXRF);
    decimal_t oldXF=(decimal_t)oldX;
    decimal_t xDiff=oldXRF-oldXF;
    decimal_t xDiffR=1.0-xDiff;

    // Each thread is responsible for a single pixel in the new image

    uint32_t c00,c01,c10,c11;
    int xLim=origImgWidth-1;
    int yLim=origImgHeight-1;

    const int currentLineOffset= oldY*origImgWidth;
    const int lineBelowOffset=(oldY==yLim?oldY:oldY+1)*origImgWidth;
    int oldXPlusOne=(oldX==xLim?oldX:oldX+1);

    c00=origImgData[currentLineOffset+oldX];
    c10=origImgData[currentLineOffset+oldXPlusOne];
    c01=origImgData[lineBelowOffset+oldX];
    c11=origImgData[lineBelowOffset+oldXPlusOne];

    decimal_t w1=xDiffR*yDiffR;
    decimal_t w2=xDiff*yDiffR;
    decimal_t w3=xDiffR*yDiff;
    decimal_t w4=xDiff*yDiff;

    uint32_t newAlpha=bilinearInterpolate(getAlpha(c00),getAlpha(c01),getAlpha(c10),getAlpha(c11),w1,w2,w3,w4);
    uint32_t newRed=bilinearInterpolate(getRed(c00),getRed(c01),getRed(c10),getRed(c11),w1,w2,w3,w4);
    uint32_t newGreen=bilinearInterpolate(getGreen(c00),getGreen(c01),getGreen(c10),getGreen(c11),w1,w2,w3,w4);
    uint32_t newBlue=bilinearInterpolate(getBlue(c00),getBlue(c01),getBlue(c10),getBlue(c11),w1,w2,w3,w4);

    newImgData[newY*newImgWidth+newX]=getColor(newAlpha,newRed,newGreen,newBlue);
}

__global__ void device_nearestNeighborResize(int origImgWidth,int newImgWidth,decimal_t origRatio,int blocksPerLine,uint32_t *origImgData,uint32_t *newImgData)
{
    // Each block is responsible for 1024 px of a single row. Do not use shared memory, as we cannot predict the distances between two lines we need in the source image.

    int threadId=threadIdx.x;
    int blockId=blockIdx.x;
    int newY=(blockId-(blockId%blocksPerLine))/blocksPerLine;
    int blockIdInLine=blockId-newY*blocksPerLine;
    int newX=blockIdInLine*CUDA_THREADS_PER_BLOCK+threadId;

    if(newX>=newImgWidth)
        return;

    int oldX=(int)round(origRatio*((decimal_t)newX));
    int oldY=(int)round(origRatio*((decimal_t)newY));

    newImgData[newY*newImgWidth+newX]=origImgData[oldY*origImgWidth+oldX];
}



uint32_t *cudaNearestNeighborResize(uint32_t *origImageData,int origImgWidth,int origImgHeight,int newImgWidth,int newImgHeight,decimal_t resizeRatio)
{
    int blocksPerLine=ceil(floatDiv(newImgWidth,CUDA_THREADS_PER_BLOCK));
    int totalNumBlocks=blocksPerLine*newImgHeight; // Each block is responsible for a single line

    if(totalNumBlocks>65535)
        return 0;

    uint32_t *device_origImageData_in;
    uint32_t *device_newImageData_out;
    size_t origImageSize=origImgWidth*origImgHeight*sizeof(uint32_t);
    size_t newImageSize=newImgWidth*newImgHeight*sizeof(uint32_t);
    hipMalloc(&device_origImageData_in,origImageSize);
    hipMalloc(&device_newImageData_out,newImageSize);

    hipMemcpy(device_origImageData_in,origImageData,origImageSize,hipMemcpyHostToDevice);

    decimal_t origRatio=decimalDiv(1.0,resizeRatio);
    device_nearestNeighborResize<<<totalNumBlocks,CUDA_THREADS_PER_BLOCK>>>(origImgWidth,newImgWidth,origRatio,blocksPerLine,device_origImageData_in,device_newImageData_out);

    uint32_t *newImageData=(uint32_t*)malloc(newImageSize);
    hipMemcpy(newImageData,device_newImageData_out,newImageSize,hipMemcpyDeviceToHost);

    hipFree(device_newImageData_out);
    hipFree(device_origImageData_in);

    return newImageData;
}

uint32_t *cudaBilinearResize(uint32_t *origImageData,int origImgWidth,int origImgHeight,int newImgWidth,int newImgHeight,decimal_t resizeRatio)
{
    int blocksPerLine=ceil(floatDiv(newImgWidth,CUDA_THREADS_PER_BLOCK));
    int totalNumBlocks=blocksPerLine*newImgHeight; // Each block is responsible for a single line

    if(totalNumBlocks>65535)
        return 0;

    uint32_t *device_origImageData_in;
    uint32_t *device_newImageData_out;
    size_t origImageSize=origImgWidth*origImgHeight*sizeof(uint32_t);
    size_t newImageSize=newImgWidth*newImgHeight*sizeof(uint32_t);
    hipMalloc(&device_origImageData_in,origImageSize);
    hipMalloc(&device_newImageData_out,newImageSize);

    hipMemcpy(device_origImageData_in,origImageData,origImageSize,hipMemcpyHostToDevice);

    decimal_t origRatio=decimalDiv(1.0,resizeRatio);
    device_bilinearResize<<<totalNumBlocks,CUDA_THREADS_PER_BLOCK>>>(origImgWidth,origImgHeight,newImgWidth,origRatio,blocksPerLine,device_origImageData_in,device_newImageData_out);

    uint32_t *newImageData=(uint32_t*)malloc(newImageSize);
    hipMemcpy(newImageData,device_newImageData_out,newImageSize,hipMemcpyDeviceToHost);

    hipFree(device_newImageData_out);
    hipFree(device_origImageData_in);

    return newImageData;
}
